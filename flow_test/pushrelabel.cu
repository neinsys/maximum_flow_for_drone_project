#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hip/device_functions.h>
#include <hip/hip_math_constants.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <queue>
#include "pushrelabel.h"


using std::vector;
using std::pair;
using edge = flowGraph::edge;
using thrust::device_vector;

#define KERNEL_CYCLES 150

#define min(a,b) (a)<(b)?(a):(b);
#define UPDIV(n, d)   (((n) + (d) - 1) / (d))

static dim3 threadsPerBlock(1024, 1, 1);


//global function
__global__ void push_relabel_kernel(edge* graph,int* startIdx, int* height, int* excessFlow, int n, int s, int t) {
	int index = blockDim.x*blockIdx.x + threadIdx.x;

	int u = index;
	int cycle = KERNEL_CYCLES;
	for (int _ = 0; _ < cycle; _++) {
		int curExcess = excessFlow[u];
		if (curExcess > 0 && height[u] < n) {
			int curLowestNeighbor = -1;
			int neighborMinHeight = (int)HIP_INF;
			
			//find lowest height in neighborhood
			int i = 0;
			int size = startIdx[u + 1] - startIdx[u];
			for(int i=0;i<size;i++){
				const edge& e = graph[startIdx[u]+i];
				int to = e.to;
				int tempHeight = height[to];
				if (neighborMinHeight > tempHeight ) {
					neighborMinHeight = tempHeight;
					curLowestNeighbor = i;
				}
				i++;
			}

			if (height[u] > neighborMinHeight) {
				int delta = min(curExcess, graph[startIdx[u]+curLowestNeighbor].cap);
				int v = graph[startIdx[u]+curLowestNeighbor].to;
				int rev = graph[startIdx[u]+curLowestNeighbor].rev;
				atomicAdd(&graph[startIdx[v]+rev].cap, delta);
				atomicSub(&graph[startIdx[u]+curLowestNeighbor].cap, delta);
				atomicAdd(&excessFlow[v], delta);
				atomicSub(&excessFlow[u], delta);
			}
			else {
				height[u] = neighborMinHeight + 1;
			}
		}
	}
}

//host function
__host__ void global_relabel_cpu(edge* graph, int* startIdx,int* height,int* excessFlow,int* excessTotal,bool* marked,int n,int t) {
	//violation-cancellation
	for (int u = 0; u < n; u++) {
		for (int i = startIdx[u]; i < startIdx[u + 1]; i++) {
			edge& e = graph[i];
			int v = e.to;
			edge& rev = graph[startIdx[v]+e.rev];
			excessFlow[u] -= e.cap;
			excessFlow[v] += e.cap;
			rev.cap += e.cap;
			e.cap = 0;
		}
	}
	//do a backwards BFS from the sink and assign the height function with each vertex's BFS tree level
	std::queue<int> que;
	que.push(t);
	std::fill(height, height + n, n);
	height[t] = 0;

	while (!que.empty()) {
		int idx = que.front();
		que.pop();

		for (int i = startIdx[idx]; i < startIdx[idx + 1]; i++) {
			edge& e = graph[i];
			int v = e.to;
			int cap = graph[startIdx[v] + e.rev].cap;
			if (cap > 0 && height[v] > height[idx] + 1) {
				height[v] = height[idx] + 1;
				que.push(v);
			}
		}
	}

	//marking vertex that not are relabeled
	for (int i = 0; i < n; i++) {
		if (!marked[i] && height[i] == n) {
			marked[i] = true;
			*excessTotal -= excessFlow[i];
		}
	}
}

//host function
__host__ void init_flow(flowGraph* graph, int* height, int* excessFlow, int* excessTotal, int n, int s) {
	for (int i = 0; i < n; i++) {
		height[i] = 0;
		excessFlow[i] = 0;
	}
	height[s] = n;

	for (edge& ori : graph->Graph[s]) {
		int to = ori.to;
		edge& rev = graph->Graph[to][ori.rev];
		int cap = ori.cap;

		ori.cap -= cap;
		rev.cap += cap;
		excessFlow[to] += cap;
		*excessTotal += cap;
	}
}


std::pair<flowGraph,int> push_relabel_cuda(flowGraph graph) {
	int n = graph.Graph.size();
	int source = graph.source;
	int sink = graph.sink;
	
	//CPU main memory initialize
	int* excessFlow_h = (int*)malloc(sizeof(int)*n);
	int* height_h = (int*)malloc(sizeof(int)*n);
	int* startIdx_h = (int*)malloc(sizeof(int)*(n+1));
	edge* graph_h;
	bool* marked = (bool*)malloc(sizeof(bool)*n);
	int excessTotal = 0;

	//CUDA global memory initialize
	int* excessFlow_d;
	int* height_d;
	edge* graph_d;
	int* startIdx_d;
	
	hipMalloc((void**)&excessFlow_d, sizeof(int)*n);
	hipMalloc((void**)&height_d, sizeof(int)*n);
	hipMalloc((void**)&startIdx_d, sizeof(int)*(n+1));

	int sum = 0;

	startIdx_h[0] = 0;
	for (int i = 0; i < n; i++) {
		sum += graph.Graph[i].size();
		startIdx_h[i + 1] = sum;
	}
	hipMemcpy(startIdx_d, startIdx_h, sizeof(int)*(n + 1), hipMemcpyHostToDevice);

	graph_h = (edge*)malloc(sizeof(edge)*sum);
	hipMalloc((void**)&graph_d, sizeof(edge*)*sum);

	//Initialize e,h,cf and excessTotal
	init_flow(&graph,height_h,excessFlow_h,&excessTotal,n,source);

	//copy e and cf from the CPU main memory to the CUDA global memory
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < graph.Graph[i].size(); j++) {
			graph_h[startIdx_h[i] + j] = graph.Graph[i][j];
		}
	}
	hipMemcpy(graph_d, graph_h, sizeof(edge)*sum, hipMemcpyHostToDevice);
	hipMemcpy(excessFlow_d, excessFlow_h, sizeof(int)*n, hipMemcpyHostToDevice);

	while (excessFlow_h[source] + excessFlow_h[sink] < excessTotal) {
		//copy h from the CPU main memory to the CUDA global memory
		hipMemcpy(height_d, height_h, sizeof(int)*n, hipMemcpyHostToDevice);

		//call push_relabel_kernel()
		int numBlocks = UPDIV(n, threadsPerBlock.x);
		push_relabel_kernel <<<numBlocks, threadsPerBlock>>> (graph_d, startIdx_d, height_d, excessFlow_d, n, source, sink);

		//copy cf, h and e from CUDA global memory to CPU main memory
		for (int i = 0; i < n; i++) {
			hipMemcpy(graph_h + startIdx_h[i], graph_d + startIdx_h[i], startIdx_h[i + 1] - startIdx_h[i], hipMemcpyDeviceToHost);
		}
		hipMemcpy(height_h, height_d, sizeof(int)*n, hipMemcpyDeviceToHost);
		hipMemcpy(excessFlow_h, excessFlow_d, sizeof(int)*n, hipMemcpyDeviceToHost);

		//call global_relabel_cpu()
		global_relabel_cpu(graph_h,startIdx_h,height_h,excessFlow_h,&excessTotal,marked,n,sink);
	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < startIdx_h[i + 1] - startIdx_h[i]; j++) {
			graph.Graph[i][j] = graph_h[startIdx_h[i] + j];
		}
	}

	return { graph,-excessFlow_h[sink] };
}