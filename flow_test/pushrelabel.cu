#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hip/device_functions.h>
#include <hip/hip_math_constants.h>
#include <queue>
#include "pushrelabel.h"


using std::vector;
using std::pair;
using edge = flowGraph::edge;

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
	bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s at %s:%d\n",
			hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#else
#define cudaCheckError(ans) ans
#endif



#define KERNEL_CYCLES 150

#define min(a,b) (a)<(b)?(a):(b);
#define UPDIV(n, d)   (((n) + (d) - 1) / (d))

static dim3 threadsPerBlock(1024, 1, 1);


//global function
__global__ void push_relabel_kernel(edge* graph, int* startIdx, int* height, int* excessFlow, int n, int s, int t) {
	int index = blockDim.x*blockIdx.x + threadIdx.x;

	int u = index;
	if ((u < 0 || u >= n) || u == t || u == s)return;

	int cycle = KERNEL_CYCLES;
	for (int _ = 0; _ < cycle; _++) {
		int curExcess = excessFlow[u];
		if (curExcess > 0 && height[u] < n) {
			int curLowestNeighbor = -1;
			int neighborMinHeight = ((int)HIP_INF) / 2;

			//find lowest height in neighborhood
			int size = startIdx[u + 1] - startIdx[u];
			for (int i = 0; i < size; i++) {
				const edge& e = graph[startIdx[u] + i];
				int to = e.to;
				int tempHeight = height[to];
				if (neighborMinHeight > tempHeight && e.cap > 0) {
					neighborMinHeight = tempHeight;
					curLowestNeighbor = i;
				}
			}

			if (height[u] > neighborMinHeight && curLowestNeighbor != -1) {
				int delta = min(curExcess, graph[startIdx[u] + curLowestNeighbor].cap);
				int v = graph[startIdx[u] + curLowestNeighbor].to;
				int rev = graph[startIdx[u] + curLowestNeighbor].rev;
				atomicAdd(&graph[startIdx[v] + rev].cap, delta);
				atomicSub(&graph[startIdx[u] + curLowestNeighbor].cap, delta);
				atomicAdd(&excessFlow[v], delta);
				atomicSub(&excessFlow[u], delta);
			}
			else {
				height[u] = neighborMinHeight + 1;
			}
		}
	}
}

//host function
__host__ void global_relabel_cpu(edge* graph, int* startIdx, int* height, int* excessFlow, int* excessTotal, bool* marked, int n, int s, int t) {
	//violation-cancellation
	for (int u = 0; u < n; u++) {
		for (int i = startIdx[u]; i < startIdx[u + 1]; i++) {
			edge& e = graph[i];
			int v = e.to;
			edge& rev = graph[startIdx[v] + e.rev];
			if (height[u] > height[v] + 1) {
				excessFlow[u] -= e.cap;
				excessFlow[v] += e.cap;
				rev.cap += e.cap;
				e.cap = 0;
			}
		}
	}
	//do a backwards BFS from the sink and assign the height function with each vertex's BFS tree level
	std::queue<int> que;
	que.push(t);
	std::fill(height, height + n, n);
	height[t] = 0;

	while (!que.empty()) {
		int idx = que.front();
		que.pop();

		for (int i = startIdx[idx]; i < startIdx[idx + 1]; i++) {
			edge& e = graph[i];
			int v = e.to;
			int cap = graph[startIdx[v] + e.rev].cap;
			if (cap > 0 && height[v] > height[idx] + 1) {
				height[v] = height[idx] + 1;
				que.push(v);
			}
		}
	}

	//marking vertex that not are relabeled
	for (int i = 0; i < n; i++) {
		if (!marked[i] && height[i] == n) {
			marked[i] = true;
			*excessTotal -= excessFlow[i];
			excessFlow[i] = 0;
		}
	}
}

//host function
__host__ void init_flow(flowGraph* graph, int* height, int* excessFlow, int* excessTotal,bool* marked, int n, int s, int t) {
	for (int i = 0; i < n; i++) {
		height[i] = 0;
		excessFlow[i] = 0;
		marked[i] = 0;
	}
	height[s] = n;

	for (edge& ori : graph->Graph[s]) {
		int to = ori.to;
		edge& rev = graph->Graph[to][ori.rev];
		int cap = ori.cap;

		ori.cap -= cap;
		rev.cap += cap;
		excessFlow[to] += cap;
		*excessTotal += cap;
	}
}


std::pair<flowGraph, int> push_relabel_cuda(flowGraph graph) {
	int n = graph.Graph.size();
	int source = graph.source;
	int sink = graph.sink;

	//CPU main memory initialize
	int* excessFlow_h = (int*)malloc(sizeof(int)*n);
	int* height_h = (int*)malloc(sizeof(int)*n);
	int* startIdx_h = (int*)malloc(sizeof(int)*(n + 1));
	edge* graph_h;
	bool* marked = (bool*)malloc(sizeof(bool)*n);
	int excessTotal = 0;

	//CUDA global memory initialize
	int* excessFlow_d;
	int* height_d;
	edge* graph_d;
	int* startIdx_d;

	cudaCheckError(hipMalloc((void**)&excessFlow_d, sizeof(int)*n));
	cudaCheckError(hipMalloc((void**)&height_d, sizeof(int)*n));
	cudaCheckError(hipMalloc((void**)&startIdx_d, sizeof(int)*(n + 1)));

	int sum = 0;

	startIdx_h[0] = 0;
	for (int i = 0; i < n; i++) {
		sum += graph.Graph[i].size();
		startIdx_h[i + 1] = sum;
	}
	cudaCheckError(hipMemcpy(startIdx_d, startIdx_h, sizeof(int)*(n + 1), hipMemcpyHostToDevice));

	graph_h = (edge*)malloc(sizeof(edge)*sum);
	cudaCheckError(hipMalloc((void**)&graph_d, sizeof(edge)*sum));

	//Initialize e,h,cf and excessTotal
	init_flow(&graph, height_h, excessFlow_h, &excessTotal, marked, n, source, sink);

	//copy e and cf from the CPU main memory to the CUDA global memory
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < graph.Graph[i].size(); j++) {
			graph_h[startIdx_h[i] + j] = graph.Graph[i][j];
		}
	}
	cudaCheckError(hipMemcpy(graph_d, graph_h, sizeof(edge)*sum, hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(excessFlow_d, excessFlow_h, sizeof(int)*n, hipMemcpyHostToDevice));

	while (excessFlow_h[source] + excessFlow_h[sink] < excessTotal) {
		//while(netFlowOutS_h!=netFlowInT_h){
			//copy h from the CPU main memory to the CUDA global memory
		hipMemcpy(height_d, height_h, sizeof(int)*n, hipMemcpyHostToDevice);

		//call push_relabel_kernel()
		int numBlocks = UPDIV(n, threadsPerBlock.x);
		push_relabel_kernel <<<numBlocks, threadsPerBlock >>> (graph_d, startIdx_d, height_d, excessFlow_d, n, source, sink);

		//copy cf, h and e from CUDA global memory to CPU main memory
		cudaCheckError(hipMemcpy(graph_h, graph_d, sizeof(edge)*sum, hipMemcpyDeviceToHost));
		cudaCheckError(hipMemcpy(height_h, height_d, sizeof(int)*n, hipMemcpyDeviceToHost));
		cudaCheckError(hipMemcpy(excessFlow_h, excessFlow_d, sizeof(int)*n, hipMemcpyDeviceToHost));

		//call global_relabel_cpu()
		global_relabel_cpu(graph_h, startIdx_h, height_h, excessFlow_h, &excessTotal, marked, n, source, sink);

	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < startIdx_h[i + 1] - startIdx_h[i]; j++) {
			graph.Graph[i][j].cap = graph_h[startIdx_h[i] + j].cap;
		}
	}

	return { graph,excessFlow_h[sink] };
}